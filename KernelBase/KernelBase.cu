#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void KernelBase(int *d_a){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    d_a[idx] = 200 * blockIdx.x + threadIdx.x;
    return ; 
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc((void **)&d_a, memSize);

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    KernelBase<<< dimGrid,dimBlock >>>(d_a);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy((void *)h_a, (const void *)d_a,memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i < numBlocks; ++i)
    {
        for (int j = 0; j < numThreadsPerBlock; ++j)
        {
            assert(h_a[i * numThreadsPerBlock + j] == 200 * i + j);
        }
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
