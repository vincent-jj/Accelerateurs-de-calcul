
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

void checkCUDAError(const char *msg);

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer and dimension for host memory
    int n, dimA;
    float *h_a;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    dimA = 8;
    h_a = (float *) malloc(dimA*sizeof(float));
    for (n=0; n<dimA; n++)
    {
        h_a[n] = (float) n;
    }

    // Part 1 of 5: allocate device memory
    size_t memSize = dimA*sizeof(float);
    hipMalloc((void**)&d_a,memSize);
    hipMalloc((void**)&d_b, memSize);

    // Part 2 of 5: host to device memory copy
    hipMemcpy((void*)d_a,(const void *)h_a,memSize, hipMemcpyHostToDevice);

    // Part 3 of 5: device to device memory copy
    hipMemcpy((void*)d_b,(const void *)d_a,memSize, hipMemcpyDeviceToDevice);

    // clear host memory
    for (n=0; n<dimA; n++)
    {
        h_a[n] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy((void *)h_a, (const void *)d_b, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (n=0; n<dimA; n++)
    {
        assert(h_a[n] == (float) n);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree((void*)d_a);
    hipFree((void*)d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    // Bonus: be sure to use cudaFreeHost for memory allocated with cudaMallocHost
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
